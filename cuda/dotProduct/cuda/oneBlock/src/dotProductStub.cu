#include "hip/hip_runtime.h"
extern "C" 
{
  #include "dotProductStub.h"
}

__global__ void MatrixMulKernel(int * _matrixA, int * _matrixB, 
                                int * _result, int _width)
{
  int k = 0, elementA = 0, elementB = 0;
  //2D thread ID
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  //valeu store the _result element that is computed by thread
  int value = 0;
  for (k = 0; k < _width; k++)
  {
      elementA = *(_matrixA + (ty*_width + k));  //Go accross the line
      elementB = *(_matrixB + (k*_width + tx));  //Go accross the column
      value += (elementA * elementB);   //Take each element
  }
  *(_result + (_width*ty + tx)) = value;

  return;
}

void matrixMultiplication(int * _matrixA, int * _matrixB, int * _result, 
                          int _width)
{
  int size = _width * _width * sizeof(int);
  int * matrixADevice, * matrixBDevice, * resultDevice;

  //Transfer MatrixA and MatrixB to device memory
  hipMalloc((void**) &matrixADevice, size);
  hipMemcpy(matrixADevice, _matrixA, size, hipMemcpyHostToDevice);
  hipMalloc((void**) &matrixBDevice, size);
  hipMemcpy(matrixBDevice, _matrixB, size, hipMemcpyHostToDevice);

  //Allocate result on the device
  hipMalloc((void**) &resultDevice, size);

  //Invoking kernel
  dim3 dimBlock(_width, _width);
  dim3 dimGrid(1, 1);
  MatrixMulKernel<<<dimGrid, dimBlock>>>(matrixADevice, matrixBDevice, 
                                         resultDevice, _width);

  //Transfer from device to host
  hipMemcpy(_result, resultDevice, size, hipMemcpyDeviceToHost);

  //Free memory
  hipFree(matrixADevice);
  hipFree(matrixBDevice);
  hipFree(resultDevice);
}

